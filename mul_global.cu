#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE 16

__global__ void MatrixMulKernel(float *M, float *N, float *P, int Width)
{ 
    int Row = blockIdx.y * blockDim.y + threadIdx.y; 
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if( Col < Width && Row < Width) 
    {
    	float Pvalue = 0;
        for(int k = 0; k < Width; ++k) 
        {
            Pvalue += M[Row * Width + k] * N[k * Width + Col];
        }
        P[Row * Width + Col] = Pvalue;
    }
} 


void cpu_matrix_mult(float *M, float *N, float *P, int Width) {
    for (int i = 0; i < Width; ++i) 
    {
        for (int j = 0; j < Width; ++j) 
        {
            int tmp = 0.0;
            for(int k = 0; k < Width; ++k) 
            {
                tmp += M[i * Width + k] * N[k * Width + j];
            }
            P[i * Width + j] = tmp;
        }
    }
}


int main()
{
    int Width =1024;
    srand(3333);

    float *h_a=0, *h_b=0, *h_c=0, *h_cc=0;
    hipHostMalloc((void **) &h_a, sizeof(float)*Width*Width);
    hipHostMalloc((void **) &h_b, sizeof(float)*Width*Width);
    hipHostMalloc((void **) &h_c, sizeof(float)*Width*Width);
    hipHostMalloc((void **) &h_cc, sizeof(float)*Width*Width);	    

    if(h_a==0 || h_b==0 || h_c==0 || h_cc==0)
    {
	printf("No asignacion de memoria\n");
	return 1;
    }
    
    for (int i = 0; i < Width; ++i) {
        for (int j = 0; j < Width; ++j) {
            h_a[i * Width + j] = rand()%1024;
        }
    }

    for (int i = 0; i < Width; ++i) {
        for (int j = 0; j < Width; ++j) {
            h_b[i * Width + j] = rand()%1024;
        }
    }

    float gpu_time_ms, cpu_time_ms;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    float *d_a=0, *d_b=0, *d_c=0;
    hipMalloc((void **) &d_a, sizeof(float)*Width*Width);
    hipMalloc((void **) &d_b, sizeof(float)*Width*Width);
    hipMalloc((void **) &d_c, sizeof(float)*Width*Width);

    if(d_a==0 || d_b==0 || d_c==0)
    {
  		printf("No asignacion Gpu\n");
		return 1;
    }

    hipMemcpy(d_a, h_a, sizeof(float)*Width*Width, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float)*Width*Width, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
    dim3 dimGrid((int)ceil(float(Width)/dimBlock.x), (int)ceil(float(Width)/dimBlock.y));

    MatrixMulKernel<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, Width);

    hipMemcpy(h_c, d_c, sizeof(int)*Width*Width, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&gpu_time_ms, start, stop);
    printf("Tiempo transcurrido en GPU: %f ms.\n\n", gpu_time_ms);


    //CPU version
    hipEventRecord(start, 0);

    cpu_matrix_mult(h_a, h_b, h_cc, Width);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_time_ms, start, stop);
    printf("Tiempo transcurrido en CPU: %f ms.\n\n", cpu_time_ms);
	

    //Validando resultados
    int all_ok = 1;
    for (int i = 0; i < Width; ++i)
    {
        for (int j = 0; j < Width; ++j)
        {
            if(h_c[i*Width + j] != h_cc[i*Width + j])
            {
                all_ok = 0;
            }
        }
    }

    if(all_ok)
    {
        printf("Todo bien!!, speedup = %f\n", cpu_time_ms / gpu_time_ms);
    }
    else
    {
        printf("Error\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}















